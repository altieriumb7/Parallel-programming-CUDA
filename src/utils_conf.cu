#include "hip/hip_runtime.h"
#include "../lib/utils_conf.cuh"

// Function to determine the configuration based on N
Config determine_config(const unsigned long long N)
{
    ParallelSortConfig config;

    config.partition_size = PARTITION_SIZE;

    // Initialize with default values
    config.total_threads = min(N, MAXTHREADSPERBLOCK);
    config.total_blocks = 1;

    // Find the largest power of two <= total_threads
    for (unsigned long long i = config.total_threads; i >= 2; i--)
    {
        if (is_power_of_two(i))
        {
            // Adjust configuration based on power of two
            config.total_threads = i;
            config.partition_size = ceil(N / float(config.total_threads));
            config.threads_per_block = config.total_threads;
            break;
        }
    }

    // If total_threads is less than WARPSIZE, adjust the values
    if (config.total_threads < WARPSIZE)
    {
        config.threads_per_block = WARPSIZE;
        config.total_threads = WARPSIZE;
        config.total_blocks = 1;
        config.partition_size = ceil(N / float(config.total_threads));
    }

    // If N is greater than the starting partition size
    if (N > config.partition_size)
    {
        config.total_threads = ceil(N / float(config.partition_size));

        // If only one block is needed
        if (config.total_threads <= MAXTHREADSPERBLOCK)
        {
            config.total_blocks = 1;
            if (config.total_threads < WARPSIZE)
            {
                config.total_threads = WARPSIZE;
                config.threads_per_block = WARPSIZE;
            }
            else
            {
                config.threads_per_block = config.total_threads;
            }

            // Find the largest power of two <= total_threads
            for (unsigned long i = config.total_threads; i >= 2; i--)
            {
                if (is_power_of_two(i))
                {
                    // Adjust configuration based on power of two
                    config.total_threads = i;
                    config.partition_size = ceil(N / float(config.total_threads));
                    config.threads_per_block = config.total_threads;
                    break;
                }
            }
        }
        // If more than one block is needed
        else
        {
            config.threads_per_block = MAXTHREADSPERBLOCK;
            config.total_blocks = min(ceil(config.total_threads / float(config.threads_per_block)), MAXBLOCKS);
            config.total_threads = config.total_blocks * config.threads_per_block;

            // Find the largest power of two <= total_threads
            for (unsigned long i = config.total_threads; i >= 2; i--)
            {
                config.total_blocks = min(ceil(i / float(MAXTHREADSPERBLOCK)), MAXBLOCKS);
                config.total_threads = config.total_blocks * config.threads_per_block;

                if (is_power_of_two(config.total_threads))
                {
                    // Adjust configuration based on power of two
                    config.partition_size = ceil(N / float(config.total_threads));
                    break;
                }
            }
        }
    }

    // Calculate the required shared memory and maximum shared memory per block
    config.required_shared_memory = N * sizeof(unsigned short) / config.total_blocks;
    hipDeviceGetAttribute(&config.max_shared_memory_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

    return config;
}
