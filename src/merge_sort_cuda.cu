#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../lib/constants.cuh"
#include "../lib/merge_sort.cuh"

// GPU helper function for bottom-up merge
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

// GPU helper function to calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

// GPU mergesort kernel
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices;
    long middle, end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = start + (width >> 1);
        if (middle > size)
            middle = size;

        end = start + width;
        if (end > size)
            end = size;

        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

// Mergesort function
void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;

    // Allocate GPU memory
    hipMalloc((void**)&D_data, size * sizeof(long));
    hipMalloc((void**)&D_swp, size * sizeof(long));
    hipMalloc((void**)&D_threads, sizeof(dim3));
    hipMalloc((void**)&D_blocks, sizeof(dim3));

    // Copy data to GPU
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

    // Copy thread and block information to GPU
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (long width = 2; width < (size << 1); width <<= 1) {
        long slices = size / (nThreads * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    // Copy sorted data back to CPU
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(B);
    hipFree(D_threads);
    hipFree(D_blocks);
}

int isSorted(long* data, long size) {
    for (int i = 1; i < size; i++) {
        if (data[i - 1] > data[i]) {
            return 0; // Not sorted
        }
    }
    return 1; // Sorted
}