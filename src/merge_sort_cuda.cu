#include "hip/hip_runtime.h"
#include "../lib/merge_sort.cuh"

void mergesort(unsigned short *data, dim3 threadsPerBlock, dim3 blocksPerGrid, unsigned long long size)
{
    unsigned short *D_data;
    unsigned short *D_swp;

    hipMalloc((void **)&D_data, size * sizeof(unsigned short));
    hipMalloc((void **)&D_swp, size * sizeof(unsigned short));

    hipMemcpy(D_data, data, size * sizeof(unsigned short), hipMemcpyHostToDevice);

    unsigned short *A = D_data;
    unsigned short *B = D_swp;

    unsigned long long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                                  blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (unsigned long long width = 2; width < (size << 1); width <<= 1)
    {
        unsigned long long slices = size / (nThreads * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, width, slices, size);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    hipMemcpy(data, A, size * sizeof(unsigned short), hipMemcpyDeviceToHost);

    hipFree(D_data);
    hipFree(D_swp);
}

__device__ unsigned long long getIdx(dim3 threads, dim3 blocks)
{
    return threadIdx.x +
           threadIdx.y * threads.x +
           threadIdx.z * threads.x * threads.y +
           blockIdx.x * threads.x * threads.y * threads.z +
           blockIdx.y * threads.x * threads.y * threads.z * blocks.x +
           blockIdx.z * threads.x * threads.y * threads.z * blocks.x * blocks.y;
}

__global__ void gpu_mergesort(unsigned short *source, unsigned short *dest, unsigned long long width, unsigned long long slices, unsigned long long size)
{
    unsigned long long idx = getIdx(blockDim, gridDim);
    unsigned long long start = width * idx * slices,
                       middle,
                       end;

    for (unsigned long long slice = 0; slice < slices; slice++)
    {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(unsigned short *source, unsigned short *dest, unsigned long long start, unsigned long long middle, unsigned long long end)
{
    unsigned long long i = start;
    unsigned long long j = middle;
    for (unsigned long long k = start; k < end; k++)
    {
        if (i < middle && (j >= end || source[i] < source[j]))
        {
            dest[k] = source[i];
            i++;
        }
        else
        {
            dest[k] = source[j];
            j++;
        }
    }
}
