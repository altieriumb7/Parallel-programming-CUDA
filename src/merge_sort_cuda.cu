#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../lib/constants.cuh"
#include "../lib/merge_sort.cuh"

// GPU helper function for bottom-up merge
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

// GPU helper function to calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

// GPU mergesort kernel
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices;
    long middle, end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = start + (width >> 1);
        if (middle > size)
            middle = size;

        end = start + width;
        if (end > size)
            end = size;

        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

// Mergesort function
void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;

    // Allocate GPU memory
    hipMalloc((void**)&D_data, size * sizeof(long));
    hipMalloc((void**)&D_swp, size * sizeof(long));
    hipMalloc((void**)&D_threads, sizeof(dim3));
    hipMalloc((void**)&D_blocks, sizeof(dim3));

    // Copy data to GPU
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

    // Copy thread and block information to GPU
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (long width = 2; width < (size << 1); width <<= 1) {
        long slices = size / (nThreads * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    // Copy sorted data back to CPU
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(B);
    hipFree(D_threads);
    hipFree(D_blocks);
}

int isSorted(long* data, long size) {
    for (int i = 1; i < size; i++) {
        if (data[i - 1] > data[i]) {
            return 0; // Not sorted
        }
    }
    return 1; // Sorted
}

//--------------------------



// ...

// GPU helper function for bottom-up merge
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end, long* shared_mem) {
    long i = start;
    long j = middle;
    long k = start + threadIdx.x;  // Compute the starting index for this thread in shared memory
    long limit = start + blockDim.x; // Calculate the limit for the loop

    while (k < end) {
        if (i < middle && (j >= end || source[i] <= source[j])) {
            shared_mem[threadIdx.x] = source[i];
            i++;
        } else {
            shared_mem[threadIdx.x] = source[j];
            j++;
        }
        k += blockDim.x; // Increment k by the number of threads in the block
        __syncthreads(); // Ensure all threads have written to shared memory

        // Copy data from shared memory back to destination array
        if (k < limit) {
            dest[k] = shared_mem[threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have copied data back
    }
}

// GPU mergesort kernel
__global__ void gpu_mergesort_shared(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices;
    long middle, end;

    // Define shared memory buffer
    __shared__ long shared_mem[SHARED_MEM_SIZE];

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = start + (width >> 1);
        if (middle > size)
            middle = size;

        end = start + width;
        if (end > size)
            end = size;

        gpu_bottomUpMerge(source, dest, start, middle, end, shared_mem);
        start += width;
    }
}


// Mergesort function
void mergesort_shared(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;

    // Allocate GPU memory
    hipMalloc((void**)&D_data, size * sizeof(long));
    hipMalloc((void**)&D_swp, size * sizeof(long));
    hipMalloc((void**)&D_threads, sizeof(dim3));
    hipMalloc((void**)&D_blocks, sizeof(dim3));

    // Copy data to GPU
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

    // Copy thread and block information to GPU
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (long width = 2; width < (size << 1); width <<= 1) {
        long slices = size / (nThreads * width) + 1;

        gpu_mergesort_shared<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    // Copy sorted data back to CPU
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(B);
    hipFree(D_threads);
    hipFree(D_blocks);
}