#include "hip/hip_runtime.h"
#include "../lib/merge_sort.cuh"

// GPU helper function for bottom-up merge
__device__ void gpuBottomUpMerge(int* src, int* dest, unsigned long long  start, unsigned long long  middle, unsigned long long  end) {
    unsigned long long  i = start;
    unsigned long long  j = middle;
    for (unsigned long long  k = start; k < end; k++) {
        if (i < middle && (j >= end || src[i] < src[j])) {
            dest[k] = src[i];
            i++;
        } else {
            dest[k] = src[j];
            j++;
        }
    }
}

__device__ unsigned long long getThreadIndex(dim3* threads, dim3* blocks) {
    unsigned long long threadIndex = threadIdx.x;
    unsigned long long threadMultiplier = threads->x;
    unsigned long long blockMultiplier = threadMultiplier * threads->y;

    return threadIndex +
           threadIdx.y * threadMultiplier +
           threadIdx.z * (threadMultiplier *= threads->y) +
           blockIdx.x  * (threadMultiplier *= threads->z) +
           blockIdx.y  * (threadMultiplier *= blocks->z) +
           blockIdx.z  * (threadMultiplier * blocks->y);
}

// GPU mergesort kernel
__global__ void gpuMergeSort(int* source, int* destination, unsigned long long size, unsigned long long  width, unsigned long long  slices, dim3* threads, dim3* blocks) {
    unsigned long long  idx = getThreadIndex(threads, blocks);
    unsigned long long  start = width * idx * slices;
    unsigned long long  middle, end;

    for (int slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = start + (width >> 1);
        if (middle > size)
            middle = size;

        end = start + width;
        if (end > size)
            end = size;

        gpuBottomUpMerge(source, destination, start, middle, end);
        start += width;
    }
}

// Mergesort function
void mergeSort(int* data, unsigned long long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    int* deviceData;
    int* deviceSwap;
    dim3* deviceThreads;
    dim3* deviceBlocks;

    // Allocate GPU memory
    hipMalloc((void**)&deviceData, size * sizeof(int));
    hipMalloc((void**)&deviceSwap, size * sizeof(int));
    hipMalloc((void**)&deviceThreads, sizeof(dim3));
    hipMalloc((void**)&deviceBlocks, sizeof(dim3));

    // Copy data to GPU
    hipMemcpy(deviceData, data, size * sizeof(int), hipMemcpyHostToDevice);

    // Copy thread and block information to GPU
    hipMemcpy(deviceThreads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(deviceBlocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    int* A = deviceData;
    int* B = deviceSwap;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (long width = 2; width < (size << 1); width <<= 1) {
        long slices = size / (nThreads * width) + 1;

        gpuMergeSort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, deviceThreads, deviceBlocks);
        __syncthreads();

        // Swap pointers A and B
        int* temp = A;
        A = B;
        B = temp;
    }

    // Copy sorted data back to CPU
    hipMemcpy(data, A, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(B);
    hipFree(deviceThreads);
    hipFree(deviceBlocks);
}

// GPU helper function for bottom-up merge
__device__ void gpuBottomUpMergeShared(int* source, int* dest, long long start, long long  middle, long long  end, int* sharedMem) {
    long long  i = start;
    long long  j = middle;
    long long  k = start + threadIdx.x;  // Calculate the starting index for this thread in shared memory
    long long  limit = start + blockDim.x; // Calculate the limit for the loop

    while (k < end) {
        if (i < middle && (j >= end || source[i] <= source[j])) {
            sharedMem[threadIdx.x] = source[i];
            i++;
        } else {
            sharedMem[threadIdx.x] = source[j];
            j++;
        }
        k += blockDim.x; // Increment k by the number of threads in the block
        __syncthreads(); // Ensure all threads have written to shared memory

        // Copy data from shared memory back to the destination array
        if (k < limit) {
            dest[k] = sharedMem[threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have copied data back
    }
}

// GPU mergesort kernel
__global__ void gpuMergeSortShared(int* source, int* dest, unsigned long long size,unsigned long long  width,unsigned long long  slices, dim3* threads, dim3* blocks) {
    unsigned long long idx = getThreadIndex(threads, blocks);
    unsigned long long  start = width * idx * slices;
    unsigned long long  middle, end;

    // Define shared memory buffer
    __shared__ int sharedMem[SHARED_MEM_SIZE];

    for (int slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = start + (width >> 1);
        if (middle > size)
            middle = size;

        end = start + width;
        if (end > size)
            end = size;

        gpuBottomUpMergeShared(source, dest, start, middle, end, sharedMem);
        start += width;
    }
}

// Mergesort function
void mergeSortShared(int* data, unsigned long long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    int* deviceData;
    int* deviceSwap;
    dim3* deviceThreads;
    dim3* deviceBlocks;

    // Allocate GPU memory
    hipMalloc((void**)&deviceData, size * sizeof(int));
    hipMalloc((void**)&deviceSwap, size * sizeof(int));
    hipMalloc((void**)&deviceThreads, sizeof(dim3));
    hipMalloc((void**)&deviceBlocks, sizeof(dim3));

    // Copy data to GPU
    hipMemcpy(deviceData, data, size * sizeof(int), hipMemcpyHostToDevice);

    // Copy thread and block information to GPU
    hipMemcpy(deviceThreads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(deviceBlocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    int* A = deviceData;
    int* B = deviceSwap;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (long width = 2; width < (size << 1); width <<= 1) {
        long slices = size / (nThreads * width) + 1;

        gpuMergeSortShared<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, deviceThreads, deviceBlocks);

        // Swap pointers A and B
        int* temp = A;
        A = B;
        B = temp;
    }

    // Copy sorted data back to CPU
    hipMemcpy(data, A, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(B);
    hipFree(deviceThreads);
    hipFree(deviceBlocks);
}
