#include "hip/hip_runtime.h"
#include "mergesort.cuh"
#include <hip/hip_runtime.h>
#include "../lib/constants.cuh"
#include "../lib/merge_sort.cuh"


void mergesort(float* data, dim3 threadsPerBlock, dim3 blocksPerGrid) 
{
    float* D_data;
    float* D_swp;
    dim3* D_threads;
    dim3* D_blocks;

    hipMalloc((void**)&D_data, size * sizeof(float));
    hipMalloc((void**)&D_swp, size * sizeof(float));

    hipMemcpy(D_data, data, size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&D_threads, sizeof(dim3));
    hipMalloc((void**)&D_blocks, sizeof(dim3));

    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    float* A = D_data;
    float* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    hipMemcpy(data, A, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
}

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) 
{
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__global__ void gpu_mergesort(float* source, float* dest, long width, long slices, dim3* threads, dim3* blocks) 
{
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices, middle, end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(float* source, float* dest, long start, long middle, long end) 
{
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}
