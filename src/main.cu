#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radix_sort.cuh"
#include "../lib/utils.cuh"
#include "../lib/constants.cuh"
 int main() {
    const int arraySize = WSIZE * LOOPS;
    unsigned int hdata[arraySize],*ddata;
    const size_t size_array = arraySize* sizeof(unsigned int);
    double t_start = 0, t_stop = 0,
    hipMalloc((void **)&ddata, size_array);
    float totalTime = 0;

    srand(time(NULL));

    for (int lcount = 0; lcount < LOOPS; lcount++) {
        // Array elements have values in the range of 1024
        unsigned int range = 1U << UPPER_BIT;

        // Fill the array with random elements
        for (int i = 0; i < arraySize; i++) {
            hdata[i] = rand() % range;
        }

        hipMemcpyToSymbol(HIP_SYMBOL(ddata), hdata, arraySize * sizeof(unsigned int));

        // Execution time measurement: start the clock
        t_start = get_time();

        parallelRadix<<<1, WSIZE>>>(ddata);
        hipDeviceSynchronize();

        // Execution time measurement: stop the clock
        t_stop = get_time();

        // Calculate the execution time
        long long duration = t_start-t_stop;
        
        totalTime += duration;

        // Copy data from device to host
        hipMemcpyFromSymbol(hdata, HIP_SYMBOL(ddata), arraySize * sizeof(unsigned int));
    }

    if (isSorted(hdata, arraySize)) {
        printf("Shared memory kernel: Array is sorted correctly.\n");
    } else {
        printf("Shared memory kernel: Array is NOT sorted correctly.\n");
    }

    printf("Parallel Radix Sort:\n");
    printf("Array size = %d\n", arraySize);
    printf("Time elapsed = %g milliseconds\n", totalTime);

    return 0;
}





