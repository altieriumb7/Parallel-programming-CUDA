#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/merge_sort.cuh"
#include "../lib/utils.cuh"
#include "../lib/quick_sort.cuh"
#include "../lib/radix_sort.cuh"
#include "../lib/utilsParallelSort.cuh"


#include <hip/hip_runtime.h>

#define size 10000

int main() {
    ParallelSortConfig sort_config = determine_config(5000);

    sort_config.blockSize = dim3(sort_config.threads_per_block);
    sort_config.gridSize = dim3(sort_config.total_blocks);
    int arr[5000];
    srand(time(NULL));
    for (int i = 0; i < 5000; i++) {
        arr[i] = rand() % 10000;
    }
    int n = sizeof(arr) / sizeof(*arr);
    quickSortIterative(arr, 0, n - 1);

    int sorted = 1; // Assume it's sorted
    for (int i = 1; i < n; i++) {
        if (arr[i - 1] > arr[i]) {
            sorted = 0; // Array is not sorted
            break;
        }
    }

    if (sorted) {
        printf("Array 'arr' is sorted.\n");
    } else {
        printf("Array 'arr' is not sorted.\n");
    }

    // Add the provided code for radix sort here

    unsigned int a[1000];
    int size_a = 1000;
    srand(time(NULL));
    for (int i = 0; i < 1000; i++) {
        a[i] = rand() % 1000;
    }

    unsigned int *dev_a;
    hipMalloc(&dev_a, size_a * sizeof(unsigned int));
    hipMemcpy(dev_a, a, size_a * sizeof(unsigned int), hipMemcpyHostToDevice);
    radix_sort_shared<<<1, size_a>>>(dev_a);
    hipMemcpy(a, dev_a, size_a * sizeof(unsigned int), hipMemcpyDeviceToHost);

    sorted = 1; // Assume it's sorted
    for (int i = 1; i < size_a; i++) {
        if (a[i - 1] > a[i]) {
            sorted = 0; // Array is not sorted
            break;
        }
    }

    if (sorted) {
        printf("Array 'a' is sorted.\n");
    } else {
        printf("Array 'a' is not sorted.\n");
    }
    sort_config = determine_config(5000);

    sort_config.blockSize = dim3(sort_config.threads_per_block);
    sort_config.gridSize = dim3(sort_config.total_blocks);
    // Sort array b using radix sort
    unsigned int b[1000];
    srand(time(NULL));
    for (int i = 0; i < 1000; i++) {
        b[i] = rand() % 1000;
    }

    unsigned int *dev_b;
    hipMalloc(&dev_b, size_a * sizeof(unsigned int));
    hipMemcpy(dev_b, b, size_a * sizeof(unsigned int), hipMemcpyHostToDevice);
    radix_sort_shared<<<1, size_a>>>(dev_b);
    hipMemcpy(b, dev_b, size_a * sizeof(unsigned int), hipMemcpyDeviceToHost);

    sorted = 1; // Assume it's sorted
    for (int i = 1; i < size_a; i++) {
        if (b[i - 1] > b[i]) {
            sorted = 0; // Array is not sorted
            break;
        }
    }

    if (sorted) {
        printf("Array 'b' is sorted.\n");
    } else {
        printf("Array 'b' is not sorted.\n");
    }

    //-------------------------------------------------------------------------------------------------------------------------
    

    // Create an array of numbers (you can replace this with your input)
    long data[5000];
    long size_data = sizeof(data) / sizeof(data[0]);

    // Print unsorted data
    for (int i = 0; i < 5000; i++) {
        data[i] = rand() % 100000;
    }

    // Sort the data using mergesort
    mergesort(data, size_data, sort_config.threads_per_block, sort_config.total_blocks);

    

    // Check if the array is sorted
    if (isSorted(data, size_data)) {
        printf("Array is sorted.\n");
    } else {
        printf("Array is not sorted.\n");
    }
    //
        // Your provided mergesort code with checks
    

    long data2[5000];
    

    // Create an array of numbers (you can replace this with your input)
    size_data = sizeof(data2) / sizeof(data2[0]);

    // Print unsorted data
    for (int i = 0; i < 5000; i++) {
        data2[i] = rand() % 100000;
    }

    // Sort the data using mergesort
    mergesort_shared(data2, size_data, sort_config.threads_per_block, sort_config.total_blocks);

    

    // Check if the array is sorted
    if (isSorted(data2, size_data)) {
        printf("Array is sorted.\n");
    } else {
        printf("Array is not sorted.\n");
    }
    

}
