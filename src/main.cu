#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radix_sort.cuh"
#include "../lib/utils.cuh"
#include "../lib/constants.cuh"
 
int main() {
    const int arraySize = 10000;
    unsigned int hdata[arraySize];
    unsigned int *ddata_glob, *ddata_shared;

    float totalTime_glob = 0;
    float totalTime_shared = 0;

    for (int lcount = 0; lcount < LOOPS; lcount++) {
        // Array elements have values in the range of 1024
        unsigned int range = 1 << UPPER_BIT;

        // Fill the array with random elements
        for (int i = 0; i < arraySize; i++) {
            hdata[i] = rand() % range;
        }

        hipMalloc((void**)&ddata_glob, arraySize * sizeof(unsigned int));
        hipMalloc((void**)&ddata_shared, arraySize * sizeof(unsigned int));

        // Copy data from host to device for global memory kernel
        hipMemcpy(ddata_glob, hdata, arraySize * sizeof(unsigned int), hipMemcpyHostToDevice);

        // Copy data from host to device for shared memory kernel
        hipMemcpy(ddata_shared, hdata, arraySize * sizeof(unsigned int), hipMemcpyHostToDevice);

        // Execution time measurement for global memory kernel
        hipEvent_t start_glob, stop_glob;
        hipEventCreate(&start_glob);
        hipEventCreate(&stop_glob);
        hipEventRecord(start_glob);

        parallelRadix_glob<<<1, WSIZE>>>(ddata_glob);
        hipDeviceSynchronize();

        hipEventRecord(stop_glob);
        hipEventSynchronize(stop_glob);
        
        float elapsedTime_glob;
        hipEventElapsedTime(&elapsedTime_glob, start_glob, stop_glob);
        totalTime_glob += elapsedTime_glob;

        // Execution time measurement for shared memory kernel
        hipEvent_t start_shared, stop_shared;
        hipEventCreate(&start_shared);
        hipEventCreate(&stop_shared);
        hipEventRecord(start_shared);

        parallelRadix_shared<<<1, WSIZE>>>(ddata_shared);
        hipDeviceSynchronize();

        hipEventRecord(stop_shared);
        hipEventSynchronize(stop_shared);

        float elapsedTime_shared;
        hipEventElapsedTime(&elapsedTime_shared, start_shared, stop_shared);
        totalTime_shared += elapsedTime_shared;

        // Copy data from device to host for global memory kernel
        hipMemcpy(hdata, ddata_glob, arraySize * sizeof(unsigned int), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(ddata_glob);
        hipFree(ddata_shared);
    }

    std::cout << "Parallel Radix Sort using Global Memory:" << std::endl;
    std::cout << "Array size = " << arraySize << std::endl;
    std::cout << "Time elapsed = " << totalTime_glob << " milliseconds" << std::endl;

    std::cout << "\nParallel Radix Sort using Shared Memory:" << std::endl;
    std::cout << "Array size = " << arraySize << std::endl;
    std::cout << "Time elapsed = " << totalTime_shared << " milliseconds" << std::endl;

    return 0;
}