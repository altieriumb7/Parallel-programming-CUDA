#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radix_sort.cuh"
#include "../lib/utils.cuh"
#include "../lib/constants.cuh"
 
int main() {
    const int arraySize = 60;
    unsigned int hdata[arraySize];
    unsigned int ddata_glob[arraySize];

    unsigned int ddata_shared[arraySize];


    float totalTime_glob = 0;
    float totalTime_shared = 0;

    for (int lcount = 0; lcount < LOOPS; lcount++) {
        // Array elements have values in the range of 1024
        unsigned int range = 1 << UPPER_BIT;

        // Fill the array with random elements
        for (int i = 0; i < arraySize; i++) {
            hdata[i] = rand() % range;
        }

        hipMalloc((void**)&ddata_glob, arraySize * sizeof(unsigned int));
        hipMalloc((void**)&ddata_shared, arraySize * sizeof(unsigned int));

        // Copy data from host to device for global memory kernel
        hipMemcpy(ddata_glob, hdata, arraySize * sizeof(unsigned int), hipMemcpyHostToDevice);

        // Copy data from host to device for shared memory kernel
        hipMemcpy(ddata_shared, hdata, arraySize * sizeof(unsigned int), hipMemcpyHostToDevice);

        // Execution time measurement for global memory kernel
        hipEvent_t start_glob, stop_glob;
        hipEventCreate(&start_glob);
        hipEventCreate(&stop_glob);
        hipEventRecord(start_glob);

        parallelRadix_glob<<<1, WSIZE>>>(ddata_glob);
        hipDeviceSynchronize();

        hipEventRecord(stop_glob);
        hipEventSynchronize(stop_glob);
        
        float elapsedTime_glob;
        hipEventElapsedTime(&elapsedTime_glob, start_glob, stop_glob);
        totalTime_glob += elapsedTime_glob;
        hipMemcpy(hdata, ddata_glob, arraySize * sizeof(unsigned int), hipMemcpyDeviceToHost);

        if (isSorted(hdata, arraySize)) {
            printf("Global memory kernel: Array is sorted correctly.\n");
        } else {
            printf("Global memory kernel: Array is NOT sorted correctly.\n");
        }

        // Execution time measurement for shared memory kernel
        hipEvent_t start_shared, stop_shared;
        hipEventCreate(&start_shared);
        hipEventCreate(&stop_shared);
        hipEventRecord(start_shared);

        parallelRadix_shared<<<1, WSIZE>>>(ddata_shared);
        hipDeviceSynchronize();

        hipEventRecord(stop_shared);
        hipEventSynchronize(stop_shared);

        float elapsedTime_shared;
        hipEventElapsedTime(&elapsedTime_shared, start_shared, stop_shared);
        totalTime_shared += elapsedTime_shared;

        // Copy data from device to host for global memory kernel
        hipMemcpy(hdata, ddata_shared, arraySize * sizeof(unsigned int), hipMemcpyDeviceToHost);
        if (isSorted(ddata_shared, arraySize)) {
            printf("Shared memory kernel: Array is sorted correctly.\n");
        } else {
            printf("Shared memory kernel: Array is NOT sorted correctly.\n");
        }
        // Free device memory
        hipFree(ddata_glob);
        hipFree(ddata_shared);
    }
    printf("Parallel Radix Sort using Global Memory:\n");
    printf("Array size = %d\n", arraySize);
    printf("Time elapsed = %f milliseconds\n", totalTime_glob);

    printf("\nParallel Radix Sort using Shared Memory:\n");
    printf("Array size = %d\n", arraySize);
    printf("Time elapsed = %f milliseconds\n", totalTime_shared);


    return 0;
}