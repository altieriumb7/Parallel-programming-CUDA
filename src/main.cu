#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radix_sort.cuh"
#include "../lib/utils.cuh"
#include "../lib/constants.cuh"
int main() {
    const int arraySize = WSIZE * LOOPS;
    //unsigned int hdata[arraySize],ddata[arraySize];
    double t_start = 0, t_stop = 0;
    unsigned int *hdata,*ddata;
    float totalTime = 0;
    const size_t size_array = arraySize * sizeof(unsigned int);
    hdata = (unsigned int *)malloc(size_array);
    hipMalloc((void **)&ddata, size_array);

    for (int lcount = 0; lcount < LOOPS; lcount++) {
        // Array elements have values in the range of 1024
        unsigned int range = 1U << UPPER_BIT;

        // Fill the array with random elements
        for (int i = 0; i < arraySize; i++) {
            hdata[i] = rand() % range;
        }

        hipMemcpy(ddata, hdata, size_array, hipMemcpyHostToDevice);

        // Execution time measurement: start the clock
        t_start = get_time();

        parallelRadix<<<1, WSIZE>>>(ddata);
        hipDeviceSynchronize();

        // Execution time measurement: stop the clock
        t_stop = get_time();

        // Calculate the execution time
        long long duration = t_start-t_stop;
        
        totalTime += duration;

        // Copy data from device to host
        hipMemcpy(ddata, hdata, size_array, hipMemcpyDeviceToHost);
    }

    if (isSorted(hdata, arraySize)) {
        printf("Shared memory kernel: Array is sorted correctly.\n");
    } else {
        printf("Shared memory kernel: Array is NOT sorted correctly.\n");
    }

    printf("Parallel Radix Sort:\n");
    printf("Array size = %d\n", arraySize);
    printf("Time elapsed = %g milliseconds\n", totalTime);

    return 0;
}





