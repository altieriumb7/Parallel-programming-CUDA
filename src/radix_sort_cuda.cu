#include "hip/hip_runtime.h"

#include "../lib/radix_sort.cuh"




__global__ void radix_sort(unsigned int *values)
{
    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
}

__device__ int plus_scan(unsigned int *x)
{
    unsigned int i = threadIdx.x; // id of thread executing this instance
    unsigned int n = blockDim.x;  // total number of threads in this block
    unsigned int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        unsigned int t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]

        __syncthreads();
    }
    return x[i];
}
__device__ void partition_by_bit(unsigned int *values, unsigned int bit)
{
    unsigned int i = threadIdx.x;
    unsigned int size = blockDim.x;
    unsigned int x_i = values[i];          // value of integer at position i
    unsigned int p_i = (x_i >> bit) & 1;   // value of bit at position bit
    values[i] = p_i;  
    __syncthreads();
    unsigned int T_before = plus_scan(values);
    unsigned int T_total  = values[size-1];
    unsigned int F_total  = size - T_total;
    __syncthreads();
    if ( p_i )
        values[T_before-1 + F_total] = x_i;
    else
        values[i - T_before] = x_i;
}

//-------------------------------------

__global__ void radix_sort_shared(unsigned int *values)
{
    int  bit;
    __shared__ unsigned int sharedValues[SHARED_MEM_SIZE];  // Define shared memory for values
    unsigned int* sValues = sharedValues;

    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit_shared(sValues, bit);  // Use shared memory for partitioning
        __syncthreads();
        
        // Copy the results back to global memory
        values[blockIdx.x * blockDim.x + threadIdx.x] = sValues[threadIdx.x];
        __syncthreads();
    }
}

__device__ int plus_scan_shared(unsigned int *x)
{
    unsigned int i = threadIdx.x; // id of thread executing this instance
    unsigned int n = blockDim.x;  // total number of threads in this block
    unsigned int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        unsigned int t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i];      // i.e., x[i] = x[i] + x[i-1]

        __syncthreads();
    }
    return x[i];
}

__device__ void partition_by_bit_shared(unsigned int *values, unsigned int bit)
{
    unsigned int i = threadIdx.x;
    unsigned int size = blockDim.x;
    unsigned int x_i = values[i];          // value of integer at position i
    unsigned int p_i = (x_i >> bit) & 1;   // value of bit at position bit
    values[i] = p_i;  
    __syncthreads();
    unsigned int T_before = plus_scan_shared(values);
    unsigned int T_total  = values[size-1];
    unsigned int F_total  = size - T_total;
    __syncthreads();
    if ( p_i )
        values[T_before-1 + F_total] = x_i;
    else
        values[i - T_before] = x_i;
}
